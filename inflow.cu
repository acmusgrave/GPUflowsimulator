#include "hip/hip_runtime.h"
#include<thrust/device_vector.h>
#include<math.h>
#include<vector>
#include<iostream>
#include "gpu_flow_sim.h"

int main(void)
{
	int nsteps =  1500;
	int plotevery = 500;
	int updatedtevery = 10;
	int n = 2000;
	float dx = 0.01;
	float L = n*dx;
	float sigma = 0.1;
	
	std::vector<float> Q0(n);
	std::vector<float> A0(n);
	std::vector<float> S(n);
	std::vector<float> r(n);
	
	for(int i=0; i<n; i++)
	{
		Q0[i] = 2;
		A0[i] = 1;
		S[i] = 0;
		r[i] = (1/(sigma*sqrtf(2*M_PI)))*exp(-0.5*pow((i*dx - L/2)/sigma, 2));
	}
	
	gpu_flow_sim::gpu_flow_sim simulator(Q0, A0, S, r, dx);
	
	simulator.set_dt();
	
	for (int i = 0; i<nsteps; i++)
	{
		if (i%plotevery == 0) simulator.output();
		if (i%updatedtevery == 0) simulator.set_dt();
		
		simulator.timestep();
	}
	simulator.output();
	
	return 0;
}
