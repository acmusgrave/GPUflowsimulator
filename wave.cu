#include "hip/hip_runtime.h"
#include<thrust/device_vector.h>
#include<math.h>
#include<vector>
#include<iostream>
#include "gpu_flow_sim.h"

int main(void)
{
	int nsteps =  300;
	int plotevery = 100;
	int updatedtevery = 10;
	int n = 2000;
	float dx = 0.01;
	float L = n*dx;
	
	std::vector<float> Q0(n);
	std::vector<float> A0(n);
	std::vector<float> S(n);
	std::vector<float> r(n);
	
	for(int i=0; i<n; i++)
	{
		Q0[i] = 0;
		A0[i] = 1 + 0.2*exp(-5*(i*dx - L/2)*(i*dx - L/2));
		S[i] = 0;
		r[i] = 0;
	}
	
	gpu_flow_sim::gpu_flow_sim simulator(Q0, A0, S, r, dx);
	
	simulator.set_dt();
	
	for (int i = 0; i<nsteps; i++)
	{
		if (i%plotevery == 0) simulator.output();
		if (i%updatedtevery == 0) simulator.set_dt();
		
		simulator.timestep();
	}
	simulator.output();
	
	return 0;
}
