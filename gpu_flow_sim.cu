#include "hip/hip_runtime.h"
#include<thrust/for_each.h>
#include<thrust/iterator/zip_iterator.h>
#include<thrust/device_vector.h>
#include<thrust/extrema.h>
#include<thrust/reduce.h>
#include<vector>
#include<iostream>
#include<fstream>
#include<stdio.h>
#include<math.h>
#include "gpu_flow_sim.h"

struct update_functor
{
	const float dt;
	const float dx;
	
	update_functor(float _dx, float _dt) : dx(_dx), dt(_dt) {}
	
    template <typename Tuple>
    __host__ __device__
    void operator()(Tuple t)
    {
		
		//unpack inputs from tuple
		float Qprev = thrust::get<0>(t);
		float Qcurr = thrust::get<1>(t);
		float Qnext = thrust::get<2>(t);
		float Aprev = thrust::get<3>(t);
		float Acurr = thrust::get<4>(t);
		float Anext = thrust::get<5>(t);
		float S = thrust::get<6>(t);
		float r = thrust::get<7>(t);
		
		//interpolate values at j+1/2, j-1/2
		float Qpos = (Qcurr + Qnext)/2;
		float Qneg = (Qcurr + Qprev)/2;
		float Apos = (Acurr + Anext)/2;
		float Aneg = (Acurr + Aprev)/2;
		
		
		//calculate eigenvalues
		float lambda1pos = Qpos/Apos - sqrtf(Apos);
		float lambda2pos = Qpos/Apos + sqrtf(Apos);
		float lambda1neg = Qneg/Aneg - sqrtf(Aneg);
		float lambda2neg = Qneg/Aneg + sqrtf(Aneg);
		
		float sgnlambda1pos = 1;
		if (lambda1pos < 0) sgnlambda1pos = -1;
		
		float sgnlambda2pos = 1;
		if (lambda2pos < 0) sgnlambda2pos = -1;
		
		float sgnlambda1neg = 1;
		if (lambda1neg < 0) sgnlambda1neg = -1;
		
		float sgnlambda2neg = 1;
		if (lambda2neg <0) sgnlambda2neg = -1;
		
		//entries for P matrix
		float Ppos11 =lambda1pos/sqrtf(1 + pow(lambda1pos, 2));
		float Ppos12 = lambda2pos/sqrtf(1 + pow(lambda2pos, 2));
		float Ppos21 = 1/sqrtf(1 + pow(lambda1pos, 2));
		float Ppos22 = 1/sqrtf(1 + pow(lambda2pos, 2));
		
		float Pneg11 =lambda1neg/sqrtf(1 + pow(lambda1neg, 2));
		float Pneg12 = lambda2neg/sqrtf(1 + pow(lambda2neg, 2));
		float Pneg21 = 1/sqrtf(1 + pow(lambda1neg, 2));
		float Pneg22 = 1/sqrtf(1 + pow(lambda2neg, 2));
		
		//inverse P matrix
		float Pinvpos11 = Ppos22/(Ppos11*Ppos22 - Ppos12*Ppos21);
		float Pinvpos12 = -Ppos12/(Ppos11*Ppos22 - Ppos12*Ppos21);
		float Pinvpos21 = -Ppos21/(Ppos11*Ppos22 - Ppos12*Ppos21);
		float Pinvpos22 = Ppos11/(Ppos11*Ppos22 - Ppos12*Ppos21);
		
		float Pinvneg11 = Pneg22/(Pneg11*Pneg22 - Pneg12*Pneg21);
		float Pinvneg12 = -Pneg12/(Pneg11*Pneg22 - Pneg12*Pneg21);
		float Pinvneg21 = -Pneg21/(Pneg11*Pneg22 - Pneg12*Pneg21);
		float Pinvneg22 = Pneg11/(Pneg11*Pneg22 - Pneg12*Pneg21);
		
		float Fprev1 = (Aprev*Aprev/2 + Qprev*Qprev/Aprev);
		float Fprev2 = Qprev;
		float Fcurr1 = (Acurr*Acurr/2 + Qcurr*Qcurr/Acurr);
		float Fcurr2 = Qcurr;
		float Fnext1 = (Anext*Anext/2 + Qnext*Qnext/Anext);
		float Fnext2 = Qnext;
		
		
		float Qterm2 = Ppos11*sgnlambda1pos*(Pinvpos11*(Fnext1-Fcurr1)/2 + Pinvpos12*(Fnext2-Fcurr2)/2) + Ppos12*sgnlambda2pos*(Pinvpos21*(Fnext1-Fcurr1)/2 + Pinvpos22*(Fnext2-Fcurr2)/2);
		float Aterm2 = Ppos21*sgnlambda1pos*(Pinvpos11*(Fnext1-Fcurr1)/2 + Pinvpos12*(Fnext2-Fcurr2)/2) + Ppos22*sgnlambda2pos*(Pinvpos21*(Fnext1-Fcurr1)/2 + Pinvpos22*(Fnext2-Fcurr2)/2);
		
		float Qterm4 = Pneg11*sgnlambda1neg*(Pinvneg11*(Fcurr1-Fprev1)/2 + Pinvneg12*(Fcurr2-Fprev2)/2) + Pneg12*sgnlambda2neg*(Pinvneg21*(Fcurr1-Fprev1)/2 + Pinvneg22*(Fcurr2-Fprev2)/2);
		float Aterm4 = Pneg21*sgnlambda1neg*(Pinvneg11*(Fcurr1-Fprev1)/2 + Pinvneg12*(Fcurr2-Fprev2)/2) + Pneg22*sgnlambda2neg*(Pinvneg21*(Fcurr1-Fprev1)/2 + Pinvneg22*(Fcurr2-Fprev2)/2);
		
		
		//output updated values to tuple
		thrust::get<8>(t) = Qcurr - (dt/dx)*((Fcurr1+Fnext1)/2 - Qterm2 - (Fcurr1+Fprev1)/2 + Qterm4) + Acurr*S*dt;
		thrust::get<9>(t) = Acurr - (dt/dx)*((Fcurr2+Fnext2)/2 - Aterm2 - (Fcurr2+Fprev2)/2 + Aterm4) + r*dt;
		
		
    }
};


// returns the larger of the magnitudes of the two eigenvalues at each node
struct max_eig_functor
{
	__host__ __device__
	float operator()(const float& Q, const float& A) const
	{ 
		float maglambda1 = fabs(Q/A - sqrtf(A));
		float maglambda2 = fabs(Q/A + sqrtf(A));
		
		if (maglambda1 > maglambda2) return maglambda1;
		return maglambda2;
	}
};


gpu_flow_sim::gpu_flow_sim(std::vector<float> Q0, std::vector<float> A0, std::vector<float> S, std::vector<float> r, float dx)
{
	this->Q = thrust::device_vector<float>(Q0);
	this->A = thrust::device_vector<float>(A0);
	this->S = thrust::device_vector<float>(S);
	this->r = thrust::device_vector<float>(r);
	this->dx = dx;
	this->n = S.size();
	//this->dt = 0.0001;
	this->set_dt();
	this->t = 0;
}

void gpu_flow_sim::timestep()
{
	
	thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(this->Q.begin(), this->Q.begin()+1, this->Q.begin()+2,    //Qprev, Qcurr, Qnext
																	this->A.begin(), this->A.begin()+1, this->A.begin()+2,  //Aprev, Acurr, Anext
																	this->S.begin()+1, this->r.begin()+1, 					//S, r
																	this->Q.begin()+1, this->A.begin()+1)),					//newQ, newA
                     thrust::make_zip_iterator(thrust::make_tuple(this->Q.end()-2, this->Q.end()-1,   this->Q.end(), 
																	this->A.end()-2, this->A.end()-1,   this->A.end(),																	
																	this->S.end()-1, this->r.end()-1,
																	this->Q.end()-1,     this->A.end()-1)),
                     update_functor(this->dx, this->dt)); // functor which implements finite volume for square channel 
	
	this->t += this->dt;
	
	return;
}

void gpu_flow_sim::output()
{
	int t_rounded_ms = int(this->t*1000 + 0.5);
	char filename[50];
	sprintf(filename, "out_%d_ms.csv", t_rounded_ms);
	std::ofstream myfile;
	myfile.open(filename);

	myfile << "X,Q,A" << std::endl;
	
    for(int i=0;i<this->n; i++)	
      {
      myfile << i*this->dx  << "," << this->Q[i] <<  "," << this->A[i] << std::endl;
      }
    myfile.close();
	return;
}

float gpu_flow_sim::set_dt()
{
	thrust::device_vector<float> maxlambda(n);
	
	thrust::transform(this->Q.begin(), this->Q.end(), this->A.begin(), maxlambda.begin(), max_eig_functor());
	
	float maxc =  *thrust::max_element(maxlambda.begin(), maxlambda.end());
	
	float cfl = 0.8;
	
	this->dt = cfl*this->dx/maxc;
	
	return this->dt;
	
}
	

